
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

void suma_vectores_gpu(float* pA, float* pB, float* pC, const int cN) {
	int idX_ = blockIdx.x * blockDim.x + threadIdx.x ; // La formula para atacar memorias = blockIdx.x * blockDim.x + threadIdx.x (la x es para porque es unidimensional (viene de un struct interno)).
	pC[idX_] = pA[idX_] + pB[idX_];
}

void suma_vectores(float* pA, float* pB, float* pC, const int cN) { // Funcion que suma vectores.
	for (unsigned int i = 0; i < cN; i++) {
		pC[i] = pA[i] + pB[i];
	}
}

int main(void) {
	const int kNumElemets = 25600;
	const size_t kNumBytes = kNumElemets * sizeof(float);

	// PASO 0: Seleccionar el dispositivo (tarjeta grafica).
	hipSetDevice(0); // Tarjeta 0, primera tarjeta que usa el dispositivo.

	// PASO 1: Declaracion de memoria en la GPU.
	float* d_a_ = NULL; // Por convencion se inicializa a NULL.
	float* d_b_ = NULL; // La d_ especifica que la variable se alojará en la GPU (es un convenio en CUDA).
	float* d_c_ = NULL;

	hipMalloc((void **)&d_a_, kNumBytes); // La GPU no entiende de tipos, por ello hay que castear a void, usamos & para pasar el puntero.
	hipMalloc((void **)&d_b_, kNumBytes); // hipMalloc es igual que malloc pero en la RAM de la GPU (VRAM).
	hipMalloc((void **)&d_c_, kNumBytes); 

	float* h_a_ = (float *)malloc(kNumBytes); // La h_ especifica que la variable se alojará en la CPU (es un convenio en CUDA).
	float* h_b_ = (float *)malloc(kNumBytes); // malloc allocates the requested memory and returns a pointer to it.
	float* h_c_ = (float *)malloc(kNumBytes);

	if (h_a_ == NULL || h_b_ == NULL || h_c_ == NULL) { // Comprobamos que el ordenador tiene memoria suficiente par alojar las variables.
		std::cerr << "Fallo al reservar la memoria";
		getchar();
		exit(-1);
	}

	for (unsigned int i = 0; i < kNumElemets; i++) { // Rellenamos los vectores con datos aleatorios.
		h_a_[i] = rand() / (float)RAND_MAX;
		h_b_[i] = rand() / (float)RAND_MAX;
	}

	// PASO 2: Copia de datos CPU a GPU.
	hipMemcpy(d_a_, h_a_, kNumBytes, hipMemcpyHostToDevice); // Destino, Origen, Cantidad de bytes a copiar, flag que especifica en que direccion (de CPU a GPU).
	hipMemcpy(d_b_, h_b_, kNumBytes, hipMemcpyHostToDevice);

	// PASO 3: Lanzar kernel (ejecutar computo).
	// FORMULA PARA LA DIRECCIONACION DE LA POSICION DE MEMORIA: 

	//suma_vectores(h_a_, h_b_, h_c_, kNumElemets);

	const int thread_per_block_ = 256;
	const int blocks_per_grid_ = kNumElemets / thread_per_block_;

	dim3 tpb_(thread_per_block_, 1, 1);
	dim3 bpg_(blocks_per_grid_, 1, 1);

}
